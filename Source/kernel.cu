#include "hip/hip_runtime.h"
#include "kernel.cuh"
#include "hiprand.h"

#include <stdio.h>
#include <cstdlib>
#include <ctime>
#include <cfloat>
#include <string>

#include "Camera.h"
#include "Ray.h"
#include "BRDF.h"

#include <iostream>

#define CUDA __host__ __device__
#define HOST __host__
#define DEVICE __device__

#define NUM_THREADS 64
#define ABSORPTION 0.25

__device__ HitInfo trace(const GPU_Scene& scene, Ray ray);
__device__ Vector3f computeRadiance(const GPU_Scene& scene, Ray r, const Camera& camera, const unsigned int idx, hiprandState *state);

CUDA struct Basis {
    Vector3f x;
    Vector3f y;
    Vector3f z;
};

__device__ Vector3f uniformHemisphereSample(unsigned int idx, hiprandState *state, Vector3f n) {
    float x = hiprand_normal(&state[idx]) * 2 - 1;
    float y = hiprand_normal(&state[idx]) * 2 - 1;
    float z = hiprand_normal(&state[idx]) * 2 - 1;

    Vector3f rand(x, y, z);
    rand.normalise();

    return (dot(rand, n) > 0) ? rand : -rand;
}

__device__ Vector3f cosineHemisphereSample(unsigned int idx, hiprandState *state, Vector3f n) {
    float u1 = hiprand_uniform(&state[idx]);
    float u2 = hiprand_uniform(&state[idx]);

    float phi = 2 * PI * u2;
    float cosTheta = sqrtf(1.0 - u1);
    float sinTheta = sqrtf(1.0 - cosTheta * cosTheta);

    float x = cosf(phi) * sinTheta;
    float y = sinf(phi) * sinTheta;
    float z = cosTheta;

    Vector3f h(n.x, n.y, n.z);
    Vector3f t = h;
    if (fabsf(t.x) <= fabsf(t.y) && fabsf(t.x) <= fabsf(t.z))
        t.x = 1.0;
    else if (fabsf(t.y) <= fabsf(t.x) && fabsf(t.y) <= fabsf(t.z))
        t.y = 1.0;
    else
        t.z = 1.0;

    Vector3f b = cross(n, t);
    b.normalise();
    t = cross(b, n);

    float nx = t.x * x + b.x * y + n.x * z;
    float ny = t.y * x + b.y * y + n.y * z;
    float nz = t.z * x + b.z * y + n.z * z;

    return Vector3f(nx, ny, nz);
}

__global__ void setup_kernel(hiprandState *state) {
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    hiprand_init(0, idx, 0, &state[idx]);
}


__device__ Vector3f directIllumination(const GPU_Scene& scene, Vector3f x, HitInfo info, const unsigned int idx, hiprandState *state) {
    Vector3f Radiance;

    Material mat = scene.dev_materials[info.mesh->materialIndex];

    // Find the light
    Mesh *light = 0;
    for (unsigned int i = 0; i < scene.meshCount; i++) {
        Mesh& m = scene.dev_meshes[i];
        Material lightMat = scene.dev_materials[m.materialIndex];
        if (lightMat.emission.length() > 1) {
            light = &scene.dev_meshes[i];
            break;
        }
    }

    // Get a random sample on the light
    Vector3f sample = light->getRandomSample(idx, state);

    // create a shadow ray to the light sample
    Ray r(x, (sample - x).normalise());

    Vector3f L = r.d;
    //Vector3f H = (L + V).normalise();

    float cos = CosTheta(info.n, L);

    // Apply BRDF
    Vector3f brdf = BRDF(info.n, L, mat);

    // Scene intersection
    info = trace(scene, r);

    // If no hit was found, there will be no lighting
    if (!info.hit) {
        return Radiance;
    }

    mat = scene.dev_materials[info.mesh->materialIndex];

    x = r.o + r.d * info.t;

    float G = (cos * CosTheta(info.n, -L)) / (info.t * info.t);

    // Check if we hit the light
    if (mat.emission.length() > EPSILON) {
        Radiance = (mat.emission * brdf * G * 13650);
    }
    return Radiance;
}

// Russian Roulette
__device__ bool isAbsorbed(hiprandState* state) {
    float p = hiprand_uniform(state);

    return p < ABSORPTION;
}

__device__ HitInfo trace(const GPU_Scene& scene, Ray ray) {
    return scene.intersect(ray);
}

__device__ Vector3f computeRadiance(const GPU_Scene& scene, Ray r, const Camera& camera, const unsigned int idx, hiprandState *state) {
    Vector3f Radiance;

    Vector3f PreRadiance[30];
    HitInfo hits[30];
    Vector3f psi[30];
    int index = 0;

    float rrWeight = 1.0f / (1.0f - ABSORPTION);
    do {
        HitInfo info = trace(scene, r);
        hits[index] = info;

        if (info.hit && info.t < camera.zFar) {
            Material mat = scene.dev_materials[info.mesh->materialIndex];

            Vector3f x = r.o + r.d * info.t;

            Vector3f Ld;
            Ld += mat.emission;
            Ld += directIllumination(scene, x, info, idx, state);

            PreRadiance[index] = Ld;

            Ray newRay(x, cosineHemisphereSample(idx, state, info.n));
            r.o = newRay.o;
            r.d = newRay.d;
            psi[index] = r.d;

            if (isAbsorbed(&state[idx])) break;

            index++;
        }
        else {
            PreRadiance[index] = Vector3f(0);
            break;
        }
    } while (index < 30);

    Radiance = PreRadiance[index];
    if (index > 0) {
        for (int i = index - 1; i >= 0; i--) {
            HitInfo info = hits[i];
            Vector3f Ld = PreRadiance[i];
            Vector3f L = psi[i];
            //Vector3f V = (camPos - x).normalise();

            Material mat = scene.dev_materials[info.mesh->materialIndex];

            // Apply BRDF
            Vector3f brdf = BRDF(info.n, L, mat);

            float cos = CosTheta(info.n, L);

            Radiance = Ld + (Radiance * brdf * cos * rrWeight);
        }
    }

    return Radiance;
}

__global__ void traceKernel(Vector3f* out, const int w, const int h,
    const Camera camera, const Basis basis, const GPU_Scene scene, hiprandState *state)
{
    unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int x = idx % w;
    unsigned int y = idx / w;

    float uvx = 2 * ((float)x / w) - 1;
    float uvy = 2 * ((float)y / h) - 1;

    float aspect = (float)w / h;

    Vector3f rayO = camera.position + (basis.x * uvx) + (basis.y * uvy);
    Vector3f rayD = ((((basis.x * aspect * uvx) + (basis.y * uvy)) * 0.33135) + basis.z).normalise();
    Ray ray(rayO, rayD);

    Vector3f Radiance = computeRadiance(scene, ray, camera, idx, state);

    //Radiance *= Vector3f(2.0f) / ((Radiance / 2.0f) + 1);

    out[idx] += Radiance;
}

bool uploadMesh(Scene &scene, GPU_Scene& gpu_scene)
{
    std::vector<Mesh> hostMeshes(scene.meshCount);

    for (unsigned int i = 0; i < scene.meshCount; i++) {
        const Mesh* mesh = scene.meshes[i];
        printf("Number of vertices: %d, Number of normals: %d, Number of faces: %d\n", mesh->numVerts, mesh->numNorms, mesh->numFaces);

        Vector3f* vertices = nullptr;
        hipMalloc((void**)&vertices, mesh->numVerts * sizeof(Vector3f));
        cudaCheckError();
        hipMemcpy(vertices, mesh->vertices, mesh->numVerts * sizeof(Vector3f), hipMemcpyHostToDevice);
        cudaCheckError();

        Vector3f* normals = nullptr;
        hipMalloc((void**)&normals, mesh->numNorms * sizeof(Vector3f));
        cudaCheckError();
        hipMemcpy(normals, mesh->normals, mesh->numNorms * sizeof(Vector3f), hipMemcpyHostToDevice);
        cudaCheckError();

        Face* faces = nullptr;
        hipMalloc((void**)&faces, mesh->numFaces * sizeof(Face));
        cudaCheckError();
        hipMemcpy(faces, mesh->faces, mesh->numFaces * sizeof(Face), hipMemcpyHostToDevice);
        cudaCheckError();

        hostMeshes[i].materialIndex = mesh->materialIndex;
        hostMeshes[i].vertices = vertices;
        hostMeshes[i].normals = normals;
        hostMeshes[i].faces = faces;
        hostMeshes[i].numVerts = mesh->numVerts;
        hostMeshes[i].numNorms = mesh->numNorms;
        hostMeshes[i].numFaces = mesh->numFaces;
    }

    hipMalloc((void**)&gpu_scene.dev_meshes, scene.meshCount * sizeof(Mesh));
    cudaCheckError();
    hipMemcpy(gpu_scene.dev_meshes, &hostMeshes[0], scene.meshCount * sizeof(Mesh), hipMemcpyHostToDevice);
    cudaCheckError();
    hipMalloc((void**)&gpu_scene.dev_materials, scene.materialCount * sizeof(Material));
    cudaCheckError();
    hipMemcpy(gpu_scene.dev_materials, &scene.materials[0], scene.materialCount * sizeof(Material), hipMemcpyHostToDevice);
    cudaCheckError();

    gpu_scene.materialCount = scene.materialCount;
    gpu_scene.meshCount = scene.meshCount;

    return true;
}

void kernelInit(Size size, hiprandState** d_state) {
    uint w = size.width;
    uint h = size.height;

    unsigned int blockSize = NUM_THREADS;
    unsigned int gridSize = (w * h) / NUM_THREADS + ((w * h) % NUM_THREADS == 0 ? 0 : 1);

    hipMalloc(d_state, gridSize * blockSize * sizeof(hiprandState));

    setup_kernel << <gridSize, blockSize >> >(*d_state);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "traceKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

Error:
    return cudaStatus;
}

hipError_t trace(Vector3f** dev_out, const Camera& camera, Size size, const GPU_Scene &scene, hiprandState* d_state) {
    uint w = size.width;
    uint h = size.height;

    unsigned int blockSize = NUM_THREADS;
    unsigned int gridSize = (w * h) / NUM_THREADS + ((w * h) % NUM_THREADS == 0 ? 0 : 1);

    Vector3f cz = normalise(camera.direction);
    Vector3f cy(0, 1, 0);
    Vector3f cx = cross(cz, cy).normalise();
    cy = cross(cx, cz);
    Basis basis = { cx, cy, cz };

    // Launch a kernel on the GPU with one thread for each element.
    traceKernel << <gridSize, blockSize >> >(*dev_out, w, h, camera, basis, scene, d_state);

    //accumKernel << <gridSize, blockSize >> >(*dev_out, 512, 512, dev_out, 1);

    // Check for any errors launching the kernel
    cudaCheckError();
}

hipError_t destroy(hiprandState** d_state) {
    hipError_t cudaStatus;

    hipFree(d_state);

    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Freeing CUDA failed: %s\n", hipGetErrorString(cudaStatus));
    }

    return cudaStatus;
}
