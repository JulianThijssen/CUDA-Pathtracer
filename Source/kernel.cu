#include "hip/hip_runtime.h"
#include "kernel.cuh"
#include "hiprand.h"

#include <stdio.h>
#include <cstdlib>
#include <ctime>
#include <cfloat>
#include <string>

#include "Camera.h"
#include "Ray.h"
#include "BRDF.h"

#define CUDA __host__ __device__
#define HOST __host__
#define DEVICE __device__
#define NUM_THREADS 64
#define ABSORPTION 0.25

__device__ HitInfo trace(const GPU_Scene& scene, Ray ray);
__device__ Vector3f computeRadiance(const GPU_Scene& scene, Ray r, const Vector3f& camPos, const unsigned int idx, hiprandState *state);

CUDA struct Basis {
    Vector3f x;
    Vector3f y;
    Vector3f z;
};

float randf() {
    return static_cast <float> (rand()) / static_cast <float> (RAND_MAX);
}

__device__ Vector3f uniformHemisphereSample(unsigned int idx, hiprandState *state, Vector3f n) {
    float x = hiprand_normal(&state[idx]) * 2 - 1;
    float y = hiprand_normal(&state[idx]) * 2 - 1;
    float z = hiprand_normal(&state[idx]) * 2 - 1;

    Vector3f rand(x, y, z);
    rand.normalise();

    return (dot(rand, n) > 0) ? rand : -rand;
}

__device__ Vector3f cosineHemisphereSample(unsigned int idx, hiprandState *state, Vector3f n) {
    float u1 = hiprand_uniform(&state[idx]);
    float u2 = hiprand_uniform(&state[idx]);

    float phi = 2 * PI * u2;
    float cosTheta = sqrtf(1.0 - u1);
    float sinTheta = sqrtf(1.0 - cosTheta * cosTheta);

    float x = cosf(phi) * sinTheta;
    float y = sinf(phi) * sinTheta;
    float z = cosTheta;

    Vector3f h(n.x, n.y, n.z);
    Vector3f t = h;
    if (fabsf(t.x) <= fabsf(t.y) && fabsf(t.x) <= fabsf(t.z))
        t.x = 1.0;
    else if (fabsf(t.y) <= fabsf(t.x) && fabsf(t.y) <= fabsf(t.z))
        t.y = 1.0;
    else
        t.z = 1.0;

    Vector3f b = cross(n, t);
    b.normalise();
    t = cross(b, n);

    float nx = t.x * x + b.x * y + n.x * z;
    float ny = t.y * x + b.y * y + n.y * z;
    float nz = t.z * x + b.z * y + n.z * z;

    return Vector3f(nx, ny, nz);
}

__global__ void setup_kernel(hiprandState *state) {
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    hiprand_init(0, idx, 0, &state[idx]);
}


__device__ Vector3f directIllumination(const GPU_Scene& scene, Vector3f x, HitInfo info, const unsigned int idx, hiprandState *state) {
    Vector3f Radiance;

    Material mat = scene.dev_materials[info.mesh->materialIndex];

    // Find the light
    Mesh *light = 0;
    for (unsigned int i = 0; i < scene.meshCount; i++) {
        Mesh& m = scene.dev_meshes[i];
        Material lightMat = scene.dev_materials[m.materialIndex];
        if (lightMat.emission.length() > 1) {
            light = &scene.dev_meshes[i];
            break;
        }
    }

    // Get a random sample on the light
    Vector3f sample = light->getRandomSample(idx, state);

    // create a shadow ray to the light sample
    Ray r(x, (sample - x).normalise());

    Vector3f L = r.d;
    //Vector3f H = (L + V).normalise();

    float cos = CosTheta(info.n, L);

    // Apply BRDF
    Vector3f brdf = BRDF(info.n, L, mat);

    // Scene intersection
    info = trace(scene, r);

    // If no hit was found, there will be no lighting
    if (!info.hit) {
        return Radiance;
    }

    mat = scene.dev_materials[info.mesh->materialIndex];

    x = r.o + r.d * info.t;

    float G = (cos * CosTheta(info.n, -L)) / (info.t * info.t);

    // Check if we hit the light
    if (mat.emission.length() > EPSILON) {
        Radiance = (mat.emission * brdf * G * 13650);
    }
    return Radiance;
}

// Russian Roulette
__device__ bool isAbsorbed(hiprandState* state) {
    float p = hiprand_uniform(state);

    return p < ABSORPTION;
}

__device__ HitInfo trace(const GPU_Scene& scene, Ray ray) {
    return scene.intersect(ray);
}

__device__ Vector3f computeRadiance(const GPU_Scene& scene, Ray r, const Camera& camera, const unsigned int idx, hiprandState *state) {
    Vector3f Radiance;

    Vector3f PreRadiance[30];
    HitInfo hits[30];
    Vector3f psi[30];
    int index = 0;

    float rrWeight = 1.0f / (1.0f - ABSORPTION);
    do {
        HitInfo info = trace(scene, r);
        hits[index] = info;

        if (info.hit && info.t < camera.zFar) {
            Material mat = scene.dev_materials[info.mesh->materialIndex];

            Vector3f x = r.o + r.d * info.t;

            Vector3f Ld;
            Ld += mat.emission;
            Ld += directIllumination(scene, x, info, idx, state);

            PreRadiance[index] = Ld;

            Ray newRay(x, cosineHemisphereSample(idx, state, info.n));
            r.o = newRay.o;
            r.d = newRay.d;
            psi[index] = r.d;

            if (isAbsorbed(&state[idx])) break;

            index++;
        }
        else {
            PreRadiance[index] = Vector3f(0);
            break;
        }
    } while (index < 30);

    Radiance = PreRadiance[index];
    if (index > 0) {
        for (int i = index - 1; i >= 0; i--) {
            HitInfo info = hits[i];
            Vector3f Ld = PreRadiance[i];
            Vector3f L = psi[i];
            //Vector3f V = (camPos - x).normalise();

            Material mat = scene.dev_materials[info.mesh->materialIndex];

            // Apply BRDF
            Vector3f brdf = BRDF(info.n, L, mat);

            float cos = CosTheta(info.n, L);

            Radiance = Ld + (Radiance * brdf * cos * rrWeight);
        }
    }

    return Radiance;
}

__global__ void traceKernel(Vector3f* out, const int w, const int h,
    const Camera camera, const Basis basis, const GPU_Scene scene, hiprandState *state)
{
    unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int x = idx % w;
    unsigned int y = idx / w;

    float uvx = 2 * ((float)x / w) - 1;
    float uvy = 2 * ((float)y / h) - 1;

    float aspect = (float)w / h;

    Vector3f rayO = camera.position + (basis.x * uvx) + (basis.y * uvy);
    Vector3f rayD = ((((basis.x * aspect * uvx) + (basis.y * uvy)) * 0.33135) + basis.z).normalise();
    Ray ray(rayO, rayD);

    Vector3f Radiance = computeRadiance(scene, ray, camera, idx, state);

    //Radiance *= Vector3f(2.0f) / ((Radiance / 2.0f) + 1);

    out[idx] += Radiance;
}

hipError_t uploadMesh(Scene &scene, GPU_Scene& gpu_scene)
{
    Mesh* mesh = new Mesh[scene.meshCount];
    for (unsigned int i = 0; i < scene.meshCount; i++) {
        memcpy(&mesh[i], scene.getMesh(i), sizeof(Mesh));
    }

    hipError_t cudaStatus;

    Mesh* h_mesh = new Mesh[scene.meshCount];

    for (unsigned int i = 0; i < scene.meshCount; i++) {
        printf("Number of vertices: %d, Number of normals: %d, Number of faces: %d\n", mesh[i].numVerts, mesh[i].numNorms, mesh[i].numFaces);

        Vector3f* vertices = 0;
        cudaStatus = hipMalloc((void**)&vertices, mesh[i].numVerts * sizeof(Vector3f));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "Vertices hipMalloc failed!");
        }
        cudaStatus = hipMemcpy(vertices, mesh[i].vertices, mesh[i].numVerts * sizeof(Vector3f), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "Vertices hipMemcpy failed!");
        }

        Vector3f* normals = 0;
        cudaStatus = hipMalloc((void**)&normals, mesh[i].numNorms * sizeof(Vector3f));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "Normals hipMalloc failed!");
        }
        cudaStatus = hipMemcpy(normals, mesh[i].normals, mesh[i].numNorms * sizeof(Vector3f), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "Vertices hipMemcpy failed!");
        }

        Face* faces = 0;
        cudaStatus = hipMalloc((void**)&faces, mesh[i].numFaces * sizeof(Face));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "Faces hipMalloc failed!");
        }
        cudaStatus = hipMemcpy(faces, mesh[i].faces, mesh[i].numFaces * sizeof(Face), hipMemcpyHostToDevice);
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "Faces hipMemcpy failed!");
        }

        h_mesh[i].materialIndex = mesh[i].materialIndex;
        h_mesh[i].vertices = vertices;
        h_mesh[i].normals = normals;
        h_mesh[i].faces = faces;
        h_mesh[i].numVerts = mesh[i].numVerts;
        h_mesh[i].numNorms = mesh[i].numNorms;
        h_mesh[i].numFaces = mesh[i].numFaces;
    }

    cudaStatus = hipMalloc((void**)&gpu_scene.dev_meshes, scene.meshCount * sizeof(Mesh));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Meshes hipMalloc failed!");
    }
    cudaStatus = hipMemcpy(gpu_scene.dev_meshes, h_mesh, scene.meshCount * sizeof(Mesh), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Meshes hipMemcpy failed!");
    }
    cudaStatus = hipMalloc((void**)&gpu_scene.dev_materials, scene.materialCount * sizeof(Material));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Materials hipMalloc failed!");
    }
    cudaStatus = hipMemcpy(gpu_scene.dev_materials, &scene.materials[0], scene.materialCount * sizeof(Material), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Materials hipMemcpy failed!");
    }

    gpu_scene.materialCount = scene.materialCount;
    gpu_scene.meshCount = scene.meshCount;

    return cudaStatus;
}

hipError_t init(uint w, uint h, hiprandState** d_state) {
    hipError_t cudaStatus;

    unsigned int blockSize = NUM_THREADS;
    unsigned int gridSize = (w * h) / NUM_THREADS + ((w * h) % NUM_THREADS == 0 ? 0 : 1);

    hipMalloc(d_state, gridSize * blockSize * sizeof(hiprandState));

    setup_kernel << <gridSize, blockSize >> >(*d_state);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "traceKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

Error:
    return cudaStatus;
}

hipError_t trace(Vector3f** dev_out, const Camera& camera, uint width, uint height, const GPU_Scene &scene, hiprandState* d_state) {
    hipError_t cudaStatus;

    unsigned int blockSize = NUM_THREADS;
    unsigned int gridSize = (width * height) / NUM_THREADS + ((width * height) % NUM_THREADS == 0 ? 0 : 1);

    Vector3f cz = normalise(camera.direction);
    Vector3f cy(0, 1, 0);
    Vector3f cx = cross(cz, cy).normalise();
    cy = cross(cx, cz);
    Basis basis = { cx, cy, cz };

    // Launch a kernel on the GPU with one thread for each element.
    traceKernel << <gridSize, blockSize >> >(*dev_out, width, height, camera, basis, scene, d_state);

    //accumKernel << <gridSize, blockSize >> >(*dev_out, 512, 512, dev_out, 1);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "traceKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

Error:
    return cudaStatus;
}

hipError_t destroy(hiprandState** d_state) {
    hipError_t cudaStatus;

    hipFree(d_state);

    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "Freeing CUDA failed: %s\n", hipGetErrorString(cudaStatus));
    }

    return cudaStatus;
}
