#include "hip/hip_runtime.h"
#include "kernel.cuh"
#include "hiprand.h"

#include <stdio.h>
#include <cstdlib>
#include <ctime>
#include <cfloat>
#include <string>

#include "Ray.h"

#define PI 3.14159265
#define ONE_OVER_PI 0.318309886

#define CUDA __host__ __device__
#define HOST __host__
#define DEVICE __device__
#define CAMERA_FAR 10000
#define ITERATIONS 5
#define NUM_THREADS 64
#define EPSILON 0.001
#define ABSORPTION 0.25

CUDA struct Basis {
	Vector3f x;
	Vector3f y;
	Vector3f z;
};

float randf() {
	return static_cast <float> (rand()) / static_cast <float> (RAND_MAX);
}

__device__ Vector3f uniformHemisphereSample(unsigned int idx, hiprandState *state, Vector3f n) {
	float x = hiprand_normal(&state[idx]) * 2 - 1;
	float y = hiprand_normal(&state[idx]) * 2 - 1;
	float z = hiprand_normal(&state[idx]) * 2 - 1;

	Vector3f rand(x, y, z);
	rand.normalise();

	return (dot(rand, n) > 0) ? rand: -rand;
}

__device__ Vector3f cosineHemisphereSample(unsigned int idx, hiprandState *state, Vector3f n) {
	float u1 = hiprand_uniform(&state[idx]);
	float u2 = hiprand_uniform(&state[idx]);

	float phi = 2 * PI * u2;
	float cosTheta = sqrtf(1.0 - u1);
	float sinTheta = sqrtf(1.0 - cosTheta * cosTheta);

	float x = cosf(phi) * sinTheta;
	float y = sinf(phi) * sinTheta;
	float z = cosTheta;

	Vector3f h(n.x, n.y, n.z);
	Vector3f t = h;
	if (fabsf(t.x) <= fabsf(t.y) && fabsf(t.x) <= fabsf(t.z))
		t.x = 1.0;
	else if (fabsf(t.y) <= fabsf(t.x) && fabsf(t.y) <= fabsf(t.z))
		t.y = 1.0;
	else
		t.z = 1.0;

	Vector3f b = cross(n, t);
	b.normalise();
	t = cross(b, n);

	float nx = t.x * x + b.x * y + n.x * z;
	float ny = t.y * x + b.y * y + n.y * z;
	float nz = t.z * x + b.z * y + n.z * z;

	return Vector3f(nx, ny, nz);
}

__global__ void setup_kernel(hiprandState *state) {
	unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	hiprand_init(0, idx, 0, &state[idx]);
}

__device__ void directLighting(const unsigned int idx, const Ray &ray,
	Vector3f &rad, const Scene &scene, hiprandState *state)
{
	Vector3f reflRad = Vector3f(1, 1, 1);
	
	Ray r(ray.o, ray.d);

	// Scene intersection
	float t;
	Vector3f n;
	Mesh *mesh;
	bool hit;

	hit = scene.intersect(r, &mesh, t, n);

	// If no hit was found, there will be no lighting
	if (!hit)
		return;

	Material mat = scene.dev_materials[mesh->materialIndex];
	if (mat.emission.length() > EPSILON) {
		rad += reflRad * mat.emission;
		return;
	}

	// Find the light
	Mesh *light = 0;
	for (unsigned int i = 0; i < scene.meshCount; i++) {
		Mesh m = scene.dev_meshes[i];
		Material lightMat = scene.dev_materials[m.materialIndex];
		if (lightMat.emission.length() > 1) {
			light = &scene.dev_meshes[i];
			break;
		}
	}

	// Get a random sample on the light
	Vector3f sample = light->getRandomSample(idx, state);

	// create a shadow ray to the light sample
	r.o = r.o + r.d * t;
	r.d = (sample - r.o).normalise();

	// Apply BRDF
	float cos = dot(n, r.d);
	float brdf = 2.0f;
	reflRad *= mat.albedo * cos * brdf;
	
	// Scene intersection
	hit = scene.intersect(r, &mesh, t, n);
	
	// If no hit was found, there will be no lighting
	if (!hit) {
		return;
	}
	
	mat = scene.dev_materials[mesh->materialIndex];

	r.o = r.o + r.d * t;
	float diff = (r.o - sample).length();

	float G = (cos * dot(n, -r.d)) / (t * t);

	// Check if we hit the light
	if (mat.emission.length() > EPSILON) {
		rad += reflRad * mat.emission * G / (1.0f / 13560);
		return;
	}
}

__device__ void indirectLighting(const unsigned int idx, const Ray &ray,
	Vector3f &rad, const Scene &scene, hiprandState *state)
{
	Ray r(ray.o, ray.d);

	Vector3f reflRad = Vector3f(1, 1, 1);
	while (true) {
		float p = hiprand_uniform(&state[idx]);

		// Russian Roulette
		if (p < ABSORPTION) {
			return;
		}

		// Scene intersection
		float t;
		Vector3f n;
		Mesh *mesh;

		scene.intersect(r, &mesh, t, n);
		Material mat = scene.dev_materials[mesh->materialIndex];

		if (t < CAMERA_FAR) {
			if (mat.emission.length() > EPSILON) {
				// We hit a light, set the total radiance
				float rrWeight = 1 / (1 - ABSORPTION);
				rad += reflRad * mat.emission * rrWeight * 2.0 * PI;
				return;
			}

			// Generate new ray from intersection
			r.o = r.o + r.d * t;
			r.d = cosineHemisphereSample(idx, state, n);

			reflRad *= mat.albedo;
		}
		else {
			// The ray escaped, no contribution
			return;
		}
	}
}

__global__ void traceKernel(float* out, const int w, const int h,
	const Vector3f o, const Basis basis, const Scene scene, hiprandState *state)
{
	unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int x = idx % w;
	unsigned int y = idx / w;

	float uvx = 2 * ((float)x / w) - 1;
	float uvy = 2 * ((float)y / h) - 1;

	float aspect = (float)w / h;

	Vector3f rayO = o + (basis.x * uvx) + (basis.y * uvy);
	Vector3f rayD = ((((basis.x * aspect * uvx) + (basis.y * uvy)) * 0.33135) + basis.z).normalise();
	Ray ray(rayO, rayD);

	Vector3f rad(0, 0, 0);

	directLighting(idx, ray, rad, scene, state);
	indirectLighting(idx, ray, rad, scene, state);

	out[y * w * 3 + x * 3 + 0] += rad.x;
	out[y * w * 3 + x * 3 + 1] += rad.y;
	out[y * w * 3 + x * 3 + 2] += rad.z;
}

hipError_t uploadMesh(Scene &scene)
{
	Mesh* mesh = new Mesh[scene.meshCount];
	for (unsigned int i = 0; i < scene.meshCount; i++) {
		memcpy(&mesh[i], &scene.getMesh(i), sizeof(Mesh));
	}
	
	hipError_t cudaStatus;

	Mesh* h_mesh = new Mesh[scene.meshCount];
	
	for (unsigned int i = 0; i < scene.meshCount; i++) {
		printf("Number of vertices: %d, Number of normals: %d, Number of faces: %d\n", mesh[i].numVerts, mesh[i].numNorms, mesh[i].numFaces);
		
		Vector3f* vertices = 0;
		cudaStatus = hipMalloc((void**)&vertices, mesh[i].numVerts * sizeof(Vector3f));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "Vertices hipMalloc failed!");
		}
		cudaStatus = hipMemcpy(vertices, mesh[i].vertices, mesh[i].numVerts * sizeof(Vector3f), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "Vertices hipMemcpy failed!");
		}

		Vector3f* normals = 0;
		cudaStatus = hipMalloc((void**)&normals, mesh[i].numNorms * sizeof(Vector3f));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "Normals hipMalloc failed!");
		}
		cudaStatus = hipMemcpy(normals, mesh[i].normals, mesh[i].numNorms * sizeof(Vector3f), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "Vertices hipMemcpy failed!");
		}

		Face* faces = 0;
		cudaStatus = hipMalloc((void**)&faces, mesh[i].numFaces * sizeof(Face));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "Faces hipMalloc failed!");
		}
		cudaStatus = hipMemcpy(faces, mesh[i].faces, mesh[i].numFaces * sizeof(Face), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "Faces hipMemcpy failed!");
		}

		h_mesh[i].materialIndex = mesh[i].materialIndex;
		h_mesh[i].vertices = vertices;
		h_mesh[i].normals = normals;
		h_mesh[i].faces = faces;
		h_mesh[i].numVerts = mesh[i].numVerts;
		h_mesh[i].numNorms = mesh[i].numNorms;
		h_mesh[i].numFaces = mesh[i].numFaces;
	}

	cudaStatus = hipMalloc((void**)&scene.dev_meshes, scene.meshCount * sizeof(Mesh));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Meshes hipMalloc failed!");
	}
	cudaStatus = hipMemcpy(scene.dev_meshes, h_mesh, scene.meshCount * sizeof(Mesh), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Meshes hipMemcpy failed!");
	}
	cudaStatus = hipMalloc((void**)&scene.dev_materials, scene.materialCount * sizeof(Material));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Materials hipMalloc failed!");
	}
	cudaStatus = hipMemcpy(scene.dev_materials, &scene.materials[0], scene.materialCount * sizeof(Material), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Materials hipMemcpy failed!");
	}

	return cudaStatus;
}

hipError_t init(uint w, uint h, hiprandState** d_state) {
	hipError_t cudaStatus;

	unsigned int blockSize = NUM_THREADS;
	unsigned int gridSize = (w * h) / NUM_THREADS + ((w * h) % NUM_THREADS == 0 ? 0 : 1);

	hipMalloc(d_state, gridSize * blockSize * sizeof(hiprandState));

	setup_kernel << <gridSize, blockSize >> >(*d_state);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "traceKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

Error:
	return cudaStatus;
}

hipError_t trace(float** dev_out, const Vector3f& o, const Vector3f& d, uint width, uint height, const Scene &scene, hiprandState* d_state) {
	hipError_t cudaStatus;
	
	unsigned int blockSize = NUM_THREADS;
	unsigned int gridSize = (width * height) / NUM_THREADS + ((width * height) % NUM_THREADS == 0 ? 0 : 1);

	Vector3f cz = d;
	Vector3f cy(0, 1, 0);
	Vector3f cx = cross(cz, cy).normalise();
	cy = cross(cx, cz);
	Basis basis = { cx, cy, cz };

	// Launch a kernel on the GPU with one thread for each element.
	traceKernel << <gridSize, blockSize >> >(*dev_out, width, height, o, basis, scene, d_state);
	
	//accumKernel << <gridSize, blockSize >> >(*dev_out, 512, 512, dev_out, 1);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "traceKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

Error:
	return cudaStatus;
}

hipError_t destroy(hiprandState** d_state) {
	hipError_t cudaStatus;

	hipFree(d_state);

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Freeing CUDA failed: %s\n", hipGetErrorString(cudaStatus));
	}

	return cudaStatus;
}
