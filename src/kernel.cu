#include "hip/hip_runtime.h"
#include "kernel.cuh"
#include "hiprand.h"

#include "ModelLoader.h"

#include <stdio.h>
#include <cstdlib>
#include <ctime>
#include <cfloat>
#include <string>

#define HOST __host__
#define DEVICE __device__
#define CAMERA_FAR 10000
#define NUM_MESHES 8
#define ITERATIONS 5
#define NUM_THREADS 32
#define EPSILON 0.0001

float randf() {
	return static_cast <float> (rand()) / static_cast <float> (RAND_MAX);
}

__device__ Vector3f generateVector(hiprandState *state, unsigned int idx, Vector3f n) {
	float x = hiprand_uniform(&state[idx]) * 2 - 1;
	float y = hiprand_uniform(&state[idx]) * 2 - 1;
	float z = hiprand_uniform(&state[idx]) * 2 - 1;

	Vector3f rand(x, y, z);
	rand.normalise();

	return (dot(rand, n) > 0) ? rand: -rand;
}

__global__ void setup_kernel(hiprandState *state) {
	unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
	hiprand_init(0, idx, 0, &state[idx]);
}

__global__ void traceKernel(float* out, const int w, const int h,
	const Vector3f o, const Vector3f cx, const Vector3f cy, const Vector3f cz,
	Mesh* meshes, hiprandState *state)
{
	//int i = threadIdx.x;
	unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned int x = idx % w;
	unsigned int y = idx / w;

	float uvx = 2 * ((float)x / w) - 1;
	float uvy = 2 * ((float)y / h) - 1;

	float aspect = (float) w / h;

	Vector3f rayO = o + (cx * uvx) + (cy * uvy);
	Vector3f rayD = ((((cx * aspect * uvx) + (cy * uvy)) * 0.33135) + cz).normalise();

	Vector3f rad(1, 1, 1);

	for (int k = 0; k < ITERATIONS; k++) {
		// Scene intersection
		float min_t = CAMERA_FAR;
		Vector3f min_n(0, 0, 0);
		Mesh* mesh;

		for (int j = 0; j < NUM_MESHES; j++) {
			Vector3f n(0, 0, 0);

			float t = intersect(rayO+rayD*EPSILON, rayD, meshes[j], n);
			if (t > 0 && t < min_t) {
				min_t = t;
				min_n.set(n.x, n.y, n.z);
				mesh = &meshes[j];
			}
		}

		if (min_t < CAMERA_FAR) {
			if (mesh->emission > 0.5) {
				rad *= mesh->emission;
				break;
			}
			rayO = rayO + rayD * min_t; // Intersection point
			rayD = generateVector(state, idx, min_n);

			float cos = dot(min_n, rayD);
			Vector3f brdf = mesh->albedo * (2 * cos);
			rad *= brdf;
		}
		else {
			rad.set(0, 0, 0);
			break;
		}
		if (k == ITERATIONS - 1) {
			rad.set(0, 0, 0);
			break;
		}
	}
	out[y * w * 3 + x * 3 + 0] += rad.x;
	out[y * w * 3 + x * 3 + 1] += rad.y;
	out[y * w * 3 + x * 3 + 2] += rad.z;
}

__global__ void accumKernel(float* out, const int w, const int h, float* in, const int k) {
	
}

hipError_t uploadMesh(Mesh** meshes)
{
	Mesh* mesh = loadMesh(std::string("path"));
	
	hipError_t cudaStatus;

	Mesh* h_mesh = new Mesh[8];
	
	for (int i = 0; i < 8; i++) {
		printf("Sizes: %d, %d, %d, %d, %f\n", mesh[i].numVerts, mesh[i].numNorms, mesh[i].numFaces, i, mesh[i].emission);
		printf("%s, %s\n", mesh[i].vertices[0].str().c_str(), mesh[i].normals[0].str().c_str());
		
		Vector3f* vertices = 0;
		cudaStatus = hipMalloc((void**)&vertices, mesh[i].numVerts * sizeof(Vector3f));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "Vertices hipMalloc failed!");
		}
		cudaStatus = hipMemcpy(vertices, mesh[i].vertices, mesh[i].numVerts * sizeof(Vector3f), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "Vertices hipMemcpy failed!");
		}

		Vector3f* normals = 0;
		cudaStatus = hipMalloc((void**)&normals, mesh[i].numNorms * sizeof(Vector3f));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "Normals hipMalloc failed!");
		}
		cudaStatus = hipMemcpy(normals, mesh[i].normals, mesh[i].numNorms * sizeof(Vector3f), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "Vertices hipMemcpy failed!");
		}

		Face* faces = 0;
		cudaStatus = hipMalloc((void**)&faces, mesh[i].numFaces * sizeof(Face));
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "Faces hipMalloc failed!");
		}
		cudaStatus = hipMemcpy(faces, mesh[i].faces, mesh[i].numFaces * sizeof(Face), hipMemcpyHostToDevice);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "Faces hipMemcpy failed!");
		}

		h_mesh[i].vertices = vertices;
		h_mesh[i].normals = normals;
		h_mesh[i].faces = faces;
		h_mesh[i].numVerts = mesh[i].numVerts;
		h_mesh[i].numNorms = mesh[i].numNorms;
		h_mesh[i].numFaces = mesh[i].numFaces;
		h_mesh[i].emission = mesh[i].emission;
		h_mesh[i].albedo = mesh[i].albedo;
	}

	cudaStatus = hipMalloc((void**)meshes, 8 * sizeof(Mesh));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Meshes hipMalloc failed!");
	}
	cudaStatus = hipMemcpy(*meshes, h_mesh, 8 * sizeof(Mesh), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Meshes hipMemcpy failed!");
	}

	return cudaStatus;
}

hipError_t init(hiprandState** d_state) {
	hipError_t cudaStatus;

	unsigned int blockSize = NUM_THREADS;
	unsigned int gridSize = (512 * 512) / NUM_THREADS + ((512 * 512) % NUM_THREADS == 0 ? 0 : 1);

	hipMalloc(d_state, gridSize * blockSize * sizeof(hiprandState));

	setup_kernel << <gridSize, blockSize >> >(*d_state);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "traceKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

Error:
	return cudaStatus;
}

hipError_t trace(float** dev_out, const Vector3f& o, const Vector3f& d, uint width, uint height, Mesh* meshes, hiprandState* d_state) {
	hipError_t cudaStatus;
	
	unsigned int blockSize = NUM_THREADS;
	unsigned int gridSize = (width * height) / NUM_THREADS + ((width * height) % NUM_THREADS == 0 ? 0 : 1);

	Vector3f cz = d;
	Vector3f cy(0, 1, 0);
	Vector3f cx = cross(cz, cy).normalise();
	cy = cross(cx, cz);

	// Launch a kernel on the GPU with one thread for each element.
	traceKernel << <gridSize, blockSize >> >(*dev_out, width, height, o, cx, cy, cz, meshes, d_state);
	
	//accumKernel << <gridSize, blockSize >> >(*dev_out, 512, 512, dev_out, 1);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "traceKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

Error:
	return cudaStatus;
}

hipError_t destroy(hiprandState** d_state) {
	hipError_t cudaStatus;

	hipFree(d_state);

	return cudaStatus;
}
